#include "hip/hip_runtime.h"
#include "MemoryPool.h"
#include "../../Nutty/Nutty/cuda/cuda_helper.h"

__global__ void freeDeviceHeapMemory(void** toFree, size_t N)
{
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= N)
    {
        return;
    }

    void* ptr = toFree[id];
    if(ptr)
    {
        free(ptr);
    }

    toFree[id] = NULL;
}

__global__ void freeDeviceHeapMemory0(void* toFree)
{
    if(toFree)
    {
        free(toFree);
    }
}

__global__ void copyHeapToGlobal(void* heapMemory, void* globalMemory, size_t N)
{
    uint id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= N)
    {
        return;
    }
    byte c = ((byte*)heapMemory)[id];
    ((byte*)globalMemory)[id] = c;
}

extern "C"
{
    void FreeDeviceHeapMemory(void** toFree, size_t N)
    {
        size_t g = nutty::cuda::GetCudaGrid(N, (size_t)256);
        freeDeviceHeapMemory<<<g, 256>>>(toFree, N);
    }

    void CopyHeapToGlobal(void* heapMemory, void* globalMemory, size_t N)
    {
        size_t g = nutty::cuda::GetCudaGrid(N, (size_t)256);
        copyHeapToGlobal<<<g, 256>>>(heapMemory, globalMemory, N);
    }

    void FreeDeviceHeapMemory0(void* toFree)
    {
        freeDeviceHeapMemory0<<<1, 1>>>(toFree);
    }
}

cuDeviceHeap::cuDeviceHeap(void) : m_pDeviceMemoryPtrs(NULL), m_offset(0), m_nextOffset(0), m_size(0), m_pDeviceSizes(NULL)
{

}

cuDeviceHeap* cuDeviceHeap::GetDevPtr(void)
{
    return m_pDevPtr;
}

void cuDeviceHeap::Print(void)
{
    for(size_t i = 0; i < m_size; ++i)
    {
        size_t blockSize = GetBlockSize(i);
        void* ptr = GetBlockContent(i);
        DEBUG_OUT_A("Block 'id=%d' 'Bytes=%d' 'Address=%p'\n", i, blockSize, ptr);
    }
}

void cuDeviceHeap::Reset(void)
{
    FreeDeviceHeapMemory(m_pDeviceMemoryPtrs, m_size);
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceMemoryPtrs, 0, m_size * sizeof(void*)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceSizes, 0, m_size * sizeof(size_t)));
}

void cuDeviceHeap::Init(size_t initialSize)
{
    if(m_size)
    {
        return;
    }

    m_compactedSize = m_size = initialSize;

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMalloc(&m_pDeviceMemoryPtrs, m_size * sizeof(void*)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceMemoryPtrs, 0, m_size * sizeof(void*)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMalloc(&m_pDeviceSizes, m_size * sizeof(size_t)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceSizes, 0, m_size * sizeof(size_t)));

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMalloc(&m_pDevPtr, sizeof(cuDeviceHeap)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(m_pDevPtr, this, sizeof(cuDeviceHeap), hipMemcpyHostToDevice));
}

size_t cuDeviceHeap::GetBlockSize(size_t id)
{
    size_t size;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(&size, m_pDeviceSizes + id, sizeof(size_t), hipMemcpyDeviceToHost));
    return size;
}

void* cuDeviceHeap::GetBlockContent(size_t id)
{
    void* _ptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(&_ptr, m_pDeviceMemoryPtrs + id, sizeof(void*), hipMemcpyDeviceToHost));
    return _ptr;
}

size_t cuDeviceHeap::GetActiveBlocks(void)
{
    return m_compactedSize;
}

size_t cuDeviceHeap::GetSize(void)
{
    return m_size;
}

// void cuDeviceHeap::Prepare(size_t threadCount)
// {
//     m_offset = m_nextOffset;
//     m_nextOffset += threadCount;
// 
//     while(threadCount > (m_size + m_offset))
//     {
//         if(!Grow())
//         {
//             assert(0 && "Grow Failed.");
//         }
//     }
// }

void cuDeviceHeap::Free(void* ptr)
{
    FreeDeviceHeapMemory0(ptr);

    void** _ptr = (void**)malloc(sizeof(void*) * m_size);
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(&_ptr, m_pDeviceMemoryPtrs, sizeof(void*) * m_size, hipMemcpyDeviceToHost));

    //slow
    for(size_t i = 0; i < m_size; ++i)
    {
        if(ptr == _ptr)
        {
            CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceMemoryPtrs + i, 0, sizeof(size_t)));
            CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemset(m_pDeviceSizes + i, 0, sizeof(size_t)));
        }
    }

    free(_ptr);
}

void cuDeviceHeap::Delete(void)
{
    if(m_pDeviceSizes)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipFree(m_pDeviceSizes));
    }

    FreeDeviceHeapMemory(m_pDeviceMemoryPtrs, m_size);

    if(m_pDeviceMemoryPtrs)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipFree(m_pDeviceMemoryPtrs));
    }
    m_pDeviceMemoryPtrs = NULL;
    m_pDeviceSizes = NULL;
}

bool cuDeviceHeap::Grow(void)
{
    void** ptrptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMalloc(&ptrptr, 2 * m_size * sizeof(void*)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(ptrptr, m_pDeviceMemoryPtrs, m_size, hipMemcpyDeviceToDevice));

    size_t* ptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMalloc(&ptr, 2 * m_size * sizeof(size_t)));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(ptr, m_pDeviceSizes, m_size, hipMemcpyDeviceToDevice));

    Delete();

    m_size = 2 * m_size;

    m_pDeviceMemoryPtrs = ptrptr;
    m_pDeviceSizes = ptr;
    return true;
}

void cuDeviceHeap::Compact(void)
{
    size_t* sizes = (size_t*)malloc(m_size * sizeof(size_t));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(sizes, m_pDeviceSizes, m_size * sizeof(size_t), hipMemcpyDeviceToHost));

    void** content = (void**)malloc(m_size * sizeof(void*));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(content, m_pDeviceMemoryPtrs, m_size * sizeof(void*), hipMemcpyDeviceToHost));

    size_t nextPos = 0;

    for(size_t i = 0; i < m_size; ++i)
    {
        if(content[i])
        {
            content[nextPos] = content[i];
            sizes[nextPos] = sizes[i];
            if(nextPos < i)
            {
                content[i] = NULL;
                sizes[i] = NULL;
            }
            nextPos++;
        }
    }

    m_compactedSize = nextPos;

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(m_pDeviceSizes, sizes, m_size * sizeof(size_t), hipMemcpyHostToDevice));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipMemcpy(m_pDeviceMemoryPtrs, content, m_size * sizeof(void*), hipMemcpyHostToDevice));

    free(content);
    free(sizes);
}

cuDeviceHeap::~cuDeviceHeap(void)
{
    Delete();

    if(m_pDevPtr)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipFree(m_pDevPtr));
    }
}

