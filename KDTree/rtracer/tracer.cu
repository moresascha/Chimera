#include "hip/hip_runtime.h"
#include "RTracer.h"
#include "../kdtree.cuh"
#include "../../../Nutty/Nutty/Wrap.h"
#include "../../../Nutty/Nutty/cuda/Module.h"
#include "../../../Nutty/Nutty/cuda/Kernel.h"
#include "../../../Nutty/Nutty/Nutty.h"
#include "../../../Nutty/Nutty/Copy.h"
#include "../../../Nutty/Nutty/cuda/Stream.h"
#include "../../../Nutty/Nutty/DeviceBuffer.h"
#include "../../../Nutty/Nutty/cuda/cuda_helper.h"
#include "../../Source/chimera/Mat4.h"
#include "../../Source/chimera/Event.h"
#include "../Source/chimera/Timer.h"

#include <cutil_math.h>

class wtf_tracer : public IRTracer
{
private:
    //nutty::MappedTexturePtr<float4> m_dst;
    hipGraphicsResource_t m_dst;
    hipGraphicsResource_t m_worldPosition;
    nutty::DeviceBuffer<float3>* m_kdData;
    nutty::DeviceBuffer<AABB>* m_kdBBox;
    nutty::DeviceBuffer<Node>* m_nodes;

    uint m_width;
    uint m_height;
    void* m_linearMem;
    size_t m_pitch;

    nutty::cuModule m_module;
    nutty::cuKernel m_kernel;

    nutty::cuTexRef m_frameBufferRef;
    nutty::cuTexRef m_worldPositionsRef;

    nutty::DeviceBuffer<float> m_view;

    IKDTree* m_tree;
    int m_enable;

    chimera::util::HTimer m_timer;

public:
    wtf_tracer(IKDTree* tree);

    void VRender(void);

    int VOnRestore(uint w, uint h);

    void ReleaseSharedResources(void);

    void ToggleEnable(void);

    void Compile(void);

    double GetLastMillis(void);

    ~wtf_tracer(void);
};

wtf_tracer::wtf_tracer(IKDTree* tree) 
    : IRTracer("wtf_tracer"), m_width(800), m_height(600), m_linearMem(NULL), m_tree(tree), m_enable(TRUE), m_worldPosition(NULL), m_dst(NULL)
{
    m_view.Resize(16);
    m_kdBBox = m_tree->GetAABBs();
    m_nodes = m_tree->GetNodes();
    m_kdData = (nutty::DeviceBuffer<float3>*)m_tree->GetData();
}

double wtf_tracer::GetLastMillis(void)
{
    return m_timer.GetMillis();
}

IRTracer* createTracer(IKDTree* tree, int flags)
{
    return new wtf_tracer(tree);
}

void wtf_tracer::ToggleEnable(void)
{
    m_enable = !m_enable;
}

void wtf_tracer::ReleaseSharedResources(void)
{
    if(m_dst)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnregisterResource(m_dst));
        m_dst = NULL;
    }

    if(m_worldPosition)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnregisterResource(m_worldPosition));
        m_worldPosition = NULL;
    }
}

void wtf_tracer::VRender(void)
{
    if(!m_enable)
    {
        return;
    }

    chimera::CmGetApp()->VGetRenderer()->VPresent();

    hipDeviceSynchronize();
    m_timer.Start();

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsMapResources(1, &m_dst, m_tree->GetDefaultStream()()));
    hipArray_t ptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsSubResourceGetMappedArray((hipArray_t*)&ptr, m_dst, 0, 0));

    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsMapResources(1, &m_worldPosition, m_tree->GetDefaultStream()()));
    hipArray_t worldPosptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsSubResourceGetMappedArray((hipArray_t*)&worldPosptr, m_worldPosition, 0, 0));

    float* view = (float*)&chimera::CmGetApp()->VGetHumanView()->VGetSceneGraph()->VGetCamera()->GetIView().m_m;
    XMFLOAT3 eye = chimera::CmGetApp()->VGetHumanView()->VGetSceneGraph()->VGetCamera()->GetEyePos().m_v;

    CUDA_RT_SAFE_CALLING_SYNC(hipMemcpy(m_view.Begin()(), view, 16 * sizeof(float), hipMemcpyHostToDevice));

    m_frameBufferRef.BindToArray((hipArray_t)ptr);
    m_worldPositionsRef.BindToArray((hipArray_t)worldPosptr);

    dim3 g;
    g.x = nutty::cuda::GetCudaGrid(m_width, 16U);
    g.y = nutty::cuda::GetCudaGrid(m_height, 16U);
    g.z = 1;
    dim3 tiles;
    tiles.x = 16;
    tiles.y = 16;
    tiles.z = 1;

    uint depth = m_tree->GetCurrentDepth();
    m_kernel.SetKernelArg(4, depth);
    m_kernel.SetKernelArg(6, eye);
    m_kernel.SetKernelArg(7, m_width);
    m_kernel.SetKernelArg(8, m_height);
    m_kernel.SetDimension(g, tiles);
    
    m_kernel.Call(m_tree->GetDefaultStream()());

    CUDA_RT_SAFE_CALLING_SYNC(hipMemcpy2DToArray(ptr, 0, 0, m_linearMem, m_pitch, m_width * sizeof(float4), m_height, hipMemcpyDeviceToDevice));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnmapResources(1, &m_dst, m_tree->GetDefaultStream()()));
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnmapResources(1, &m_worldPosition, m_tree->GetDefaultStream()()));

    hipDeviceSynchronize();
    m_timer.Stop();
}

BOOL wtf_tracer::VOnRestore(UINT w, UINT h)
{
    m_width = w;
    m_height = h;

    if(m_linearMem)
    {
        hipFree(m_linearMem);
    }
    
    CUDA_RT_SAFE_CALLING_SYNC(hipMallocPitch(&m_linearMem, &m_pitch, m_width* sizeof(float4), m_height));
    
    CUDA_RT_SAFE_CALLING_SYNC(hipMemset(m_linearMem, 0, m_pitch * m_height));

    chimera::IDeviceTexture* colorBuffer = chimera::CmGetApp()->VGetRenderer()->VGetCurrentRenderTarget()->VGetTexture();
    chimera::IDeviceTexture* worldPosition = chimera::CmGetApp()->VGetRenderer()->VGetAlbedoBuffer()->VGetRenderTarget(chimera::eDiff_WorldPositionTarget)->VGetTexture();

    CUDA_RT_SAFE_CALLING_NO_SYNC(cudaGraphicsD3D11RegisterResource(&m_dst, (ID3D11Texture2D*)colorBuffer->VGetDevicePtr(), cudaGraphicsMapFlagsNone));

    CUDA_RT_SAFE_CALLING_NO_SYNC(cudaGraphicsD3D11RegisterResource(&m_worldPosition, (ID3D11Texture2D*)worldPosition->VGetDevicePtr(), cudaGraphicsMapFlagsNone));

    Compile();

    return TRUE;
}

void wtf_tracer::Compile(void)
{
    /*nutty::cuModule test;
    test.Create("ptx/tracer_kernel.ptx");*/

    m_module.Create("ptx/tracer_kernel.ptx");
    m_kernel.Create(m_module.GetFunction("simpleSphereTracer"));

    m_kernel.SetKernelArg(0, m_linearMem);
    m_kernel.SetKernelArg(1, *m_kdData);
    m_kernel.SetKernelArg(2, *m_kdBBox);
    m_kernel.SetKernelArg(3, *m_nodes);
    m_kernel.SetKernelArg(5, m_view);

    m_frameBufferRef = m_module.GetTexRef("src");
    m_frameBufferRef.NormalizedCoords();
    m_frameBufferRef.SetFilterMode(HIP_TR_FILTER_MODE_LINEAR);
    m_frameBufferRef.SetFormat(HIP_AD_FORMAT_FLOAT, 4);
    m_frameBufferRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 0);
    m_frameBufferRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 1);

    m_worldPositionsRef = m_module.GetTexRef("worldPosTexture");
    m_worldPositionsRef.NormalizedCoords();
    m_worldPositionsRef.SetFilterMode(HIP_TR_FILTER_MODE_LINEAR);
    m_worldPositionsRef.SetFormat(HIP_AD_FORMAT_FLOAT, 4);
    m_worldPositionsRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 0);
    m_worldPositionsRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 1);
}

wtf_tracer::~wtf_tracer(void)
{
    if(m_linearMem)
    {
        hipFree(m_linearMem);
    }
}