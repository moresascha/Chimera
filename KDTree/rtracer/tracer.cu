#include "hip/hip_runtime.h"
#include "tracer.cuh"
#include "../kdtree.cuh"
#include "../../../Nutty/Nutty/Wrap.h"
#include "../../../Nutty/Nutty/cuda/Module.h"
#include "../../../Nutty/Nutty/cuda/Kernel.h"
#include "../../../Nutty/Nutty/Nutty.h"
#include "../../../Nutty/Nutty/Copy.h"
#include "../../../Nutty/Nutty/Fill.h"
#include "../../../Nutty/Nutty/Scan.h"
#include "../../../Nutty/Nutty/cuda/Stream.h"
#include "../../../Nutty/Nutty/DeviceBuffer.h"
#include "../../../Nutty/Nutty/HostBuffer.h"
#include "../../../Nutty/Nutty/cuda/cuda_helper.h"
#include "../../Source/chimera/Mat4.h"
#include "../../Source/chimera/Event.h"
#include "../Source/chimera/Timer.h"
#include "../DoubleBuffer.h"

#include <cutil_math.h>

float g_sphereRadius;

class wtf_tracer : public IRTracer
{
private:
    //nutty::MappedTexturePtr<float4> m_dst;
    hipGraphicsResource_t m_dst;
    hipGraphicsResource_t m_worldPosition;
    nutty::DeviceBuffer<float3>* m_kdData;
    nutty::DeviceBuffer<BBox>* m_kdBBox;
    Node m_nodes;

    nutty::DeviceBuffer<Ray> m_rays[2];
    nutty::DeviceBuffer<uint> m_initRayMask;
    nutty::DeviceBuffer<uint> m_scannedRayMask;
    nutty::DeviceBuffer<uint> m_sums;

    nutty::DeviceBuffer<Ray> m_shadowRays[2];
    nutty::DeviceBuffer<uint> m_shadowRayMask;

    uint m_width;
    uint m_height;
    void* m_linearMem;
    size_t m_pitch;

    nutty::cuModule m_module;
    nutty::cuKernel m_kernel;
    nutty::cuKernel m_computeInitialrays;
    nutty::cuKernel m_computeRays;
    nutty::cuKernel m_computeShadowRays;

    nutty::cuTexRef m_frameBufferRef;
    nutty::cuTexRef m_worldPositionsRef;

    nutty::DeviceBuffer<float> m_view;

    IKDTree* m_tree;
    int m_enable;
    uint m_lastRayCount;
    uint m_lastShadowRaysCount;
    chimera::util::HTimer m_timer;

public:
    wtf_tracer(IKDTree* tree);

    void VRender(void);

    int VOnRestore(uint w, uint h);

    void ReleaseSharedResources(void);

    uint GetLastRayCount(void) { return m_lastRayCount; }

    uint GetLastShadowRayCount(void) { return m_lastShadowRaysCount; }

    void ToggleEnable(void);

    void Compile(void);

    double GetLastMillis(void);

    ~wtf_tracer(void);
};

wtf_tracer::wtf_tracer(IKDTree* tree) 
    : IRTracer("wtf_tracer"), m_width(800), m_height(600), m_linearMem(NULL), m_tree(tree), m_enable(TRUE), m_worldPosition(NULL), m_dst(NULL), m_lastRayCount(0), m_lastShadowRaysCount(0)
{
    m_view.Resize(16);
    m_kdBBox = m_tree->GetAABBs();
    m_nodes = m_tree->GetNodes();
    m_kdData = (nutty::DeviceBuffer<float3>*)m_tree->GetData();
    m_timer.Start();
    m_timer.Stop();
}

double wtf_tracer::GetLastMillis(void)
{
    return m_timer.GetMillis();
}

void wtf_tracer::ToggleEnable(void)
{
    m_enable = !m_enable;
}

void wtf_tracer::ReleaseSharedResources(void)
{
    if(m_dst)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnregisterResource(m_dst));
        m_dst = NULL;
    }

    if(m_worldPosition)
    {
        CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsUnregisterResource(m_worldPosition));
        m_worldPosition = NULL;
    }
}

void wtf_tracer::VRender(void)
{
    if(!m_enable)
    {
        return;
    }
    
    chimera::CmGetApp()->VGetRenderer()->VPresent();

    hipDeviceSynchronize();
    m_timer.Start();
    
    CUDA_RT_SAFE_CALLING_SYNC(hipGraphicsMapResources(1, &m_dst));
    hipArray_t ptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsSubResourceGetMappedArray((hipArray_t*)&ptr, m_dst, 0, 0));

    CUDA_RT_SAFE_CALLING_SYNC(hipGraphicsMapResources(1, &m_worldPosition));
    hipArray_t worldPosptr;
    CUDA_RT_SAFE_CALLING_NO_SYNC(hipGraphicsSubResourceGetMappedArray((hipArray_t*)&worldPosptr, m_worldPosition, 0, 0));

    float* view = (float*)&chimera::CmGetApp()->VGetHumanView()->VGetSceneGraph()->VGetCamera()->GetIView().m_m;
    XMFLOAT3 eye = chimera::CmGetApp()->VGetHumanView()->VGetSceneGraph()->VGetCamera()->GetEyePos().m_v;

    CUDA_RT_SAFE_CALLING_SYNC(hipMemcpy(m_view.Begin()(), view, 16 * sizeof(float), hipMemcpyHostToDevice));

    m_frameBufferRef.BindToArray((hipArray_t)ptr);
    m_worldPositionsRef.BindToArray((hipArray_t)worldPosptr);

    dim3 tiles;
    tiles.x = 16;
    tiles.y = 16;
    tiles.z = 1;

    dim3 g;
    g.x = nutty::cuda::GetCudaGrid(m_width, (uint)tiles.x);
    g.y = nutty::cuda::GetCudaGrid(m_height, (uint)tiles.y);
    g.z = 1;

    uint depth = m_tree->GetCurrentDepth();

#if 0
    m_kernel.SetDimension(g, tiles);
    m_kernel.SetKernelArg(4, depth);
    m_kernel.SetKernelArg(5, m_view);
    m_kernel.SetKernelArg(6, eye);
    m_kernel.SetKernelArg(7, g_sphereRadius);
    m_kernel.SetKernelArg(8, m_width);
    m_kernel.SetKernelArg(9, m_height);

    m_kernel.Call();

#else
    
    m_lastRayCount = 0;
    m_lastShadowRaysCount = 0;

    nutty::ZeroMem(m_initRayMask);
    nutty::ZeroMem(m_scannedRayMask);
    nutty::ZeroMem(m_shadowRayMask);
    nutty::ZeroMem(m_sums);

    m_computeInitialrays.SetDimension(g, tiles);

    m_computeInitialrays.SetKernelArg(4, m_rays[0]);
    m_computeInitialrays.SetKernelArg(5, m_shadowRays[0]);
    m_computeInitialrays.SetKernelArg(3, eye);
    m_computeInitialrays.SetKernelArg(8, m_width);
    m_computeInitialrays.SetKernelArg(9, m_height);
    m_computeInitialrays.Call();

    
    DEVICE_SYNC_CHECK();
    
    nutty::ExclusivePrefixSumScan(m_shadowRayMask.Begin(), m_shadowRayMask.End(), m_scannedRayMask.Begin(), m_sums.Begin());
    DEVICE_SYNC_CHECK();

    nutty::Compact(m_shadowRays[1].Begin(), m_shadowRays[0].Begin(), m_shadowRays[0].End(), m_shadowRayMask.Begin(), m_scannedRayMask.Begin(), 0U);

    DEVICE_SYNC_CHECK();

    uint shadowRaysCount = *(m_scannedRayMask.End()-1) + *(m_shadowRayMask.End()-1);

    m_lastShadowRaysCount += shadowRaysCount;

    DEVICE_SYNC_CHECK();

    if(shadowRaysCount > 0)
    {
        uint blockSize = 256;
        g = nutty::cuda::GetCudaGrid(shadowRaysCount, blockSize);
        m_computeShadowRays.SetDimension(g, blockSize);
        m_computeShadowRays.SetKernelArg(1, m_shadowRays[1]);
        m_computeShadowRays.SetKernelArg(4, depth);
        m_computeShadowRays.SetKernelArg(5, g_sphereRadius);
        m_computeShadowRays.SetKernelArg(6, m_width);
        m_computeShadowRays.SetKernelArg(7, shadowRaysCount);
        m_computeShadowRays.Call();
    }

    uint lastRayCount = m_width * m_height;
    uint recDepth = 2;
    byte toggle = 0;
    for(int i = 0; i < recDepth; ++i)
    {       
        nutty::ZeroMem(m_scannedRayMask);
        nutty::ZeroMem(m_sums);

        DEVICE_SYNC_CHECK();

        nutty::ExclusivePrefixSumScan(m_initRayMask.Begin(), m_initRayMask.Begin() + lastRayCount, m_scannedRayMask.Begin(), m_sums.Begin());
        DEVICE_SYNC_CHECK();

        nutty::Compact(m_rays[(toggle+1)%2].Begin(), m_rays[toggle].Begin(), m_rays[toggle].Begin() + lastRayCount, m_initRayMask.Begin(), m_scannedRayMask.Begin(), 0U);
        DEVICE_SYNC_CHECK();

        lastRayCount = *(m_scannedRayMask.Begin() + lastRayCount - 1) + *(m_initRayMask.Begin() + lastRayCount - 1);

        m_lastRayCount += lastRayCount;

        if(lastRayCount > 0)
        {
            uint blockSize = 256;
            g = nutty::cuda::GetCudaGrid(lastRayCount, blockSize);
            m_computeRays.SetDimension(g, blockSize);
            m_computeRays.SetKernelArg(6, m_rays[(toggle+1)%2]);
            m_computeRays.SetKernelArg(7, m_shadowRays[0]);
            m_computeRays.SetKernelArg(8, depth);
            m_computeRays.SetKernelArg(9, g_sphereRadius);
            m_computeRays.SetKernelArg(10, m_width);
            m_computeRays.SetKernelArg(11, m_height);
            m_computeRays.SetKernelArg(12, i);
            m_computeRays.SetKernelArg(13, lastRayCount);
            m_computeRays.SetKernelArg(14, eye);
            m_computeRays.SetKernelArg(15, recDepth);
            m_computeRays.Call();
   
            nutty::ZeroMem(m_scannedRayMask);
            nutty::ZeroMem(m_sums);

            DEVICE_SYNC_CHECK();

            nutty::ExclusivePrefixSumScan(m_shadowRayMask.Begin(), m_shadowRayMask.Begin() + lastRayCount, m_scannedRayMask.Begin(), m_sums.Begin());
            DEVICE_SYNC_CHECK();

            nutty::Compact(m_shadowRays[1].Begin(), m_shadowRays[0].Begin(), m_shadowRays[0].Begin() + lastRayCount, m_shadowRayMask.Begin(), m_scannedRayMask.Begin(), 0U);

            DEVICE_SYNC_CHECK();

            shadowRaysCount = *(m_scannedRayMask.Begin() + lastRayCount - 1) + *(m_shadowRayMask.Begin() + lastRayCount - 1);

            m_lastShadowRaysCount += shadowRaysCount;

            DEVICE_SYNC_CHECK();
            
            if(shadowRaysCount > 0)
            {
                g = nutty::cuda::GetCudaGrid(shadowRaysCount, blockSize);
                m_computeShadowRays.SetDimension(g, blockSize);
                m_computeShadowRays.SetKernelArg(1, m_shadowRays[1]);
                m_computeShadowRays.SetKernelArg(4, depth);
                m_computeShadowRays.SetKernelArg(5, g_sphereRadius);
                m_computeShadowRays.SetKernelArg(6, m_width);
                m_computeShadowRays.SetKernelArg(7, shadowRaysCount);
                m_computeShadowRays.Call();
            }
            toggle = (toggle + 1) % 2;
            DEVICE_SYNC_CHECK();
        }
        else
        {
            break;
        }
    }
#endif
    
    hipDeviceSynchronize();

    CUDA_RT_SAFE_CALLING_SYNC(hipMemcpy2DToArray(ptr, 0, 0, m_linearMem, m_pitch, m_width * sizeof(float4), m_height, hipMemcpyDeviceToDevice));
    CUDA_RT_SAFE_CALLING_SYNC(hipGraphicsUnmapResources(1, &m_dst));
    CUDA_RT_SAFE_CALLING_SYNC(hipGraphicsUnmapResources(1, &m_worldPosition));

    m_timer.Stop();
}

BOOL wtf_tracer::VOnRestore(UINT w, UINT h)
{
    m_width = w;
    m_height = h;

    dim3 dim;
    dim.x = nutty::cuda::GetCudaGrid(m_width, 16U);
    dim.y = nutty::cuda::GetCudaGrid(m_height, 16U);

    m_rays[0].Resize(dim.x * dim.y * 16 * 16);
    m_rays[1].Resize(dim.x * dim.y * 16 * 16);
    m_initRayMask.Resize(dim.x * dim.y * 16 * 16);
    m_scannedRayMask.Resize(dim.x * dim.y * 16 * 16);
    m_sums.Resize((m_scannedRayMask.Size()) / 512);

    m_shadowRayMask.Resize(dim.x * dim.y * 16 * 16);
    m_shadowRays[0].Resize(dim.x * dim.y * 16 * 16);
    m_shadowRays[1].Resize(dim.x * dim.y * 16 * 16);

    if(m_linearMem)
    {
        hipFree(m_linearMem);
    }
    
    CUDA_RT_SAFE_CALLING_SYNC(hipMallocPitch(&m_linearMem, &m_pitch, m_width * sizeof(float4), m_height));
    
    CUDA_RT_SAFE_CALLING_SYNC(hipMemset(m_linearMem, 0, m_pitch * m_height));

    chimera::IDeviceTexture* colorBuffer = chimera::CmGetApp()->VGetRenderer()->VGetCurrentRenderTarget()->VGetTexture();
    chimera::IDeviceTexture* worldPosition = chimera::CmGetApp()->VGetRenderer()->VGetAlbedoBuffer()->VGetRenderTarget(chimera::eDiff_WorldPositionTarget)->VGetTexture();

    CUDA_RT_SAFE_CALLING_NO_SYNC(cudaGraphicsD3D11RegisterResource(&m_dst, (ID3D11Texture2D*)colorBuffer->VGetDevicePtr(), cudaGraphicsMapFlagsNone));

    CUDA_RT_SAFE_CALLING_NO_SYNC(cudaGraphicsD3D11RegisterResource(&m_worldPosition, (ID3D11Texture2D*)worldPosition->VGetDevicePtr(), cudaGraphicsMapFlagsNone));

    Compile();

    return TRUE;
}

void wtf_tracer::Compile(void)
{
    m_module.Create("ptx/tracer_kernel.ptx");
    m_kernel.Create(m_module.GetFunction("simpleSphereTracer"));

    m_kernel.SetKernelArg(0, m_linearMem);
    m_kernel.SetKernelArg(1, *m_kdData);
    m_kernel.SetKernelArg(2, *m_kdBBox);
    m_kernel.SetKernelArg(3, m_nodes);

    m_computeInitialrays.Create(m_module.GetFunction("computeInitialRays"));
    m_computeInitialrays.SetKernelArg(0, m_linearMem);
    m_computeInitialrays.SetKernelArg(1, *m_kdBBox);
    m_computeInitialrays.SetKernelArg(2, m_view);
    m_computeInitialrays.SetKernelArg(6, m_initRayMask);
    m_computeInitialrays.SetKernelArg(7, m_shadowRayMask);

    m_computeRays.Create(m_module.GetFunction("computeRays"));
    m_computeRays.SetKernelArg(0, m_linearMem);
    m_computeRays.SetKernelArg(1, m_nodes);
    m_computeRays.SetKernelArg(2, *m_kdData);
    m_computeRays.SetKernelArg(3, m_initRayMask);
    m_computeRays.SetKernelArg(4, m_shadowRayMask);
    m_computeRays.SetKernelArg(5, *m_kdBBox);

    m_computeShadowRays.Create(m_module.GetFunction("computeShadowRays"));
    m_computeShadowRays.SetKernelArg(0, m_linearMem);
    m_computeShadowRays.SetKernelArg(2, m_nodes);
    m_computeShadowRays.SetKernelArg(3, *m_kdData);

    m_frameBufferRef = m_module.GetTexRef("src");
    m_frameBufferRef.NormalizedCoords();
    m_frameBufferRef.SetFilterMode(HIP_TR_FILTER_MODE_LINEAR);
    m_frameBufferRef.SetFormat(HIP_AD_FORMAT_FLOAT, 4);
    m_frameBufferRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 0);
    m_frameBufferRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 1);

    m_worldPositionsRef = m_module.GetTexRef("worldPosTexture");
    m_worldPositionsRef.NormalizedCoords();
    m_worldPositionsRef.SetFilterMode(HIP_TR_FILTER_MODE_LINEAR);
    m_worldPositionsRef.SetFormat(HIP_AD_FORMAT_FLOAT, 4);
    m_worldPositionsRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 0);
    m_worldPositionsRef.SetAddressMode(HIP_TR_ADDRESS_MODE_WRAP, 1);
}

wtf_tracer::~wtf_tracer(void)
{
    if(m_linearMem)
    {
        hipFree(m_linearMem);
    }
}

IRTracer* createTracer(IKDTree* tree, float radius, int flags)
{
    g_sphereRadius = radius;
    return new wtf_tracer(tree);
}
